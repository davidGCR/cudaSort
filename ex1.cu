#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <vector>
#include <fstream>
#include <iostream>
#include <stdlib.h>
#include "common/book.h"
using namespace std;


void load_file(vector<double>* vec, string fileName) {
	// string fileName = "data.txt";
	ifstream reader;
	reader.open(fileName, ios::in);
	if (reader.is_open()) {
		char linea[60];
		double valor;
		while (!reader.eof()) {
			reader >> linea;
			if (strlen(&linea[0]) != 0) {
				valor = stod(linea);
				vec->push_back(valor);
			}
		}

	}
	else {
		cout << "No se pudo abrir el fichero..." << endl;
	}
	reader.close();
}

void load_data_array(double* vec, int size,string fileName) {
	// string fileName = "data.txt";
	int count =0;
	// int size = sizeof(vec)/sizeof(*vec);
	ifstream reader;
	reader.open(fileName, ios::in);
	if (reader.is_open()) {
		char linea[60];
		double valor;
		while (!reader.eof()&&count<size) {
			reader >> linea;
			if (strlen(&linea[0]) != 0) {
				valor = stod(linea);
				vec[count] = valor;
				// cout<<"añadio: "<<vec[count]<<endl;
				count++;
				
				// vec->push_back(valor);
			}
		}

	}
	else {
		cout << "No se pudo abrir el fichero..." << endl;
	}
	reader.close();
}

const int N = 16; 
const int blocksize = 16; 
 
__global__ void hello(char *a, int *b) 
{
	a[threadIdx.x] += b[threadIdx.x];
}
////////////////////////////////////////////////////// C++ ///////////////////////////////////////////////////////////////
template <class RandomAccessIterator>
void print_vector(RandomAccessIterator inicio, RandomAccessIterator fin, string titulo) {
	RandomAccessIterator index;
	cout<<"=========="<<titulo<<"=========="<<endl;
	for (index = inicio; index != fin; index++) {
		cout << "indx: " << index - inicio << "->" << inicio[index - inicio]
			<< endl;
	}
}
void print_array(double* array, int size,string titulo) {
	cout<<"=========="<<titulo<<"=========="<<endl;
	// int size = (sizeof(array))/sizeof(*array);
	for (int  i = 0; i < size; i++) {
		cout << "indx: " << i << "->" << array[i]<< endl;
	}
}

void quickSort(vector<double>* array, int left, int right){
	// cout << "usando serial quicksort..." << endl;
	int i = left, j = right;
	double tmp;
	double pivot = (*array)[(left + right) / 2];
		/* PARTITION PART */
		while (i <= j) {
			while ((*array)[i] < pivot)
				i++;
			while ((*array)[j] > pivot)
				j--;
			if (i <= j) {
				tmp = (*array)[i];
				(*array)[i] = (*array)[j];
				(*array)[j] = tmp;
				i++;
				j--;
			}
		}
	if (left < j){ quickSort(array, left, j); }			
	if (i < right){ quickSort(array, i, right); }
}
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////777
 
void cuda_info(){
	hipDeviceProp_t  prop;

    int count;
    // HANDLE_ERROR( hipGetDeviceCount( &count ) );
	hipGetDeviceCount( &count );
    for (int i=0; i< count; i++) {
        // HANDLE_ERROR( hipGetDeviceProperties( &prop, i ) );
		hipGetDeviceProperties( &prop, i ) ;
        printf( "   --- General Information for device %d ---\n", i );
        printf( "Name:  %s\n", prop.name );
        printf( "Compute capability:  %d.%d\n", prop.major, prop.minor );
        printf( "Clock rate:  %d\n", prop.clockRate );
        printf( "Device copy overlap:  " );
        if (prop.deviceOverlap)
            printf( "Enabled\n" );
        else
            printf( "Disabled\n");
        printf( "Kernel execution timeout :  " );
        if (prop.kernelExecTimeoutEnabled)
            printf( "Enabled\n" );
        else
            printf( "Disabled\n" );

        printf( "   --- Memory Information for device %d ---\n", i );
        printf( "Total global mem:  %ld\n", prop.totalGlobalMem );
        printf( "Total constant Mem:  %ld\n", prop.totalConstMem );
        printf( "Max mem pitch:  %ld\n", prop.memPitch );
        printf( "Texture Alignment:  %ld\n", prop.textureAlignment );

        printf( "   --- MP Information for device %d ---\n", i );
        printf( "Multiprocessor count:  %d\n",
                    prop.multiProcessorCount );
        printf( "Shared mem per mp:  %ld\n", prop.sharedMemPerBlock );
        printf( "Registers per mp:  %d\n", prop.regsPerBlock );
        printf( "Threads in warp:  %d\n", prop.warpSize );
        printf( "Max threads per block:  %d\n",
                    prop.maxThreadsPerBlock );
        printf( "Max thread dimensions:  (%d, %d, %d)\n",
                    prop.maxThreadsDim[0], prop.maxThreadsDim[1],
                    prop.maxThreadsDim[2] );
        printf( "Max grid dimensions:  (%d, %d, %d)\n",
                    prop.maxGridSize[0], prop.maxGridSize[1],
                    prop.maxGridSize[2] );
        printf( "\n" );
    }
}
int main()
{
	// char a[N] = "Hello \0\0\0\0\0\0";
	// int b[N] = {15, 10, 6, 0, -11, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
 
	// char *ad;
	// int *bd;
	// const int csize = N*sizeof(char);
	// const int isize = N*sizeof(int);
 
	// printf("%s", a);
 
	// hipMalloc( (void**)&ad, csize ); 
	// hipMalloc( (void**)&bd, isize ); 
	// hipMemcpy( ad, a, csize, hipMemcpyHostToDevice ); 
	// hipMemcpy( bd, b, isize, hipMemcpyHostToDevice ); 
	
	// dim3 dimBlock( blocksize, 1 );
	// dim3 dimGrid( 1, 1 );
	// hello<<<dimGrid, dimBlock>>>(ad, bd);
	// hipMemcpy( a, ad, csize, hipMemcpyDeviceToHost ); 
	// hipFree( ad );
	// hipFree( bd );
	
	// printf("%s\n", a);
	
	// cuda_info();
	// hipDeviceProp_t  prop;
	// int nDevices=-1;
	// // hipGetDeviceCount( &count );
	// hipError_t err = hipGetDeviceCount(&nDevices);
  	// if (err != hipSuccess) printf("%s\n", hipGetErrorString(err));
	// cout<<"no_devices: "<<nDevices<<endl;
	// hipGetDeviceProperties( &prop, 1 ) ;
	// cout<<"nombre: "<< prop.name<<endl;
	
	// cuda_info();

	//host_variables

	int size = 1000;
	double* data = new double[size];
	data[50]=108;

	//load variable
	string file="1000Int.txt";
	load_data_array(data,size,file);
	// print_array(data,size,"data array raw");
	// size = *(&data + 1) - data;
	// load_file(&vec,file);
	// size =vec.size() ;
	// cout<<"size: "<<data[50]<<endl;
	// quickSort(&vec,0,vec.size()-1);
	// print_vector(vec.begin(),vec.end(),"ordenamiento secuencial");

	//device_variables
	double* d_data;

	// vector<double>* result;
	
	HANDLE_ERROR(hipMalloc( (void**)&d_data, size * sizeof(double) ));
	HANDLE_ERROR(hipMemcpy(d_data,data,size * sizeof(double),hipMemcpyHostToDevice));

	double result[size];
	HANDLE_ERROR(hipMemcpy(result,d_data,size * sizeof(double),hipMemcpyDeviceToHost));
	print_array(&result[0],size,"data array device");

	hipFree(d_data);
	
	return EXIT_SUCCESS;
}